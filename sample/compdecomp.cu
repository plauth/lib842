#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <sys/time.h>

#include <lib842/sw.h>
#include <lib842/hip/hip_runtime.h>

#define THREADS_PER_BLOCK 32
#define STRLEN 32
#define STREAM_COUNT 3
#ifdef USE_UNIFIED_MEM
#define CHUNKS_PER_THREAD 32
#else
#define CHUNKS_PER_THREAD 1024
#endif

#define CHECK_ERROR(err)                                                       \
	if (err != hipSuccess) {                                              \
		printf("Error: %s\n", hipGetErrorString(err));                \
		exit(-1);                                                      \
	}

long long timestamp()
{
	struct timeval te;
	gettimeofday(&te, NULL);
	long long ms = te.tv_sec * 1000LL + te.tv_usec / 1000;
	return ms;
}

size_t nextMultipleOfChunkSize(size_t input)
{
	size_t size =
		LIB842_CUDA_CHUNK_SIZE * CHUNKS_PER_THREAD * THREADS_PER_BLOCK;
	return (input + (size - 1)) & ~(size - 1);
}

int main(int argc, const char *argv[])
{
#ifdef LIB842_CUDA_STRICT
	printf("Running in strict mode (i.e. fully compatible to the hardware-based nx842 unit).\n");
#endif
	uint8_t *in, *compressed, *decompressed;
#ifndef USE_UNIFIED_MEM
	uint64_t *compressedD, *decompressedD;
#else
	printf("Using unified memory model.\n");
#endif
#ifdef USE_STREAMS
	printf("Using streams for overlapping memory transfers and computation.\n");
	hipStream_t streams[STREAM_COUNT];
	for (int i = 0; i < STREAM_COUNT; i++) {
		hipStreamCreate(&streams[i]);
	}
#endif
	size_t ilen, olen, dlen;
	ilen = olen = dlen = 0;
	long long timestart_comp, timeend_comp;
	long long timestart_decomp, timeend_decomp;
	hipError_t cuda_error;
	int count = 0;
	hipGetDeviceCount(&count);
	printf(" %d CUDA devices found\n", count);
	if (!count)
		::exit(EXIT_FAILURE);

	if (argc <= 1) {
		ilen = STRLEN;
		olen = ilen * 2;
		dlen = ilen;
#ifdef USE_UNIFIED_MEM
		hipMallocManaged(&in, ilen);
		hipMallocManaged(&compressed, olen);
		hipMallocManaged(&decompressed, dlen);
#else
		hipHostAlloc((void **)&in, ilen, hipHostMallocPortable);
		hipHostAlloc((void **)&compressed, olen,
			      hipHostMallocPortable);
		hipHostAlloc((void **)&decompressed, dlen,
			      hipHostMallocPortable);

		hipMalloc((void **)&compressedD, olen);
		hipMalloc((void **)&decompressedD, dlen);
		hipMemset(compressedD, 0, olen);
		hipMemset(decompressedD, 0, dlen);
#endif

		memset(in, 0, ilen);
		memset(compressed, 0, olen);
		memset(decompressed, 0, dlen);

		uint8_t tmp[] = {
			0x30, 0x30, 0x31, 0x31, 0x32, 0x32, 0x33, 0x33,
			0x34, 0x34, 0x35, 0x35, 0x36, 0x36, 0x37, 0x37,
			0x38, 0x38, 0x39, 0x39, 0x40, 0x40, 0x41, 0x41,
			0x42, 0x42, 0x43, 0x43, 0x44, 0x44, 0x45, 0x45
		}; //"0011223344556677889900AABBCCDDEE";

		memcpy(in, tmp, STRLEN);

	} else if (argc == 2) {
		FILE *fp;
		fp = fopen(argv[1], "rb");
		fseek(fp, 0, SEEK_END);
		size_t flen = (size_t)ftell(fp);
		ilen = flen;
		printf("original file length: %zu\n", ilen);
		ilen = nextMultipleOfChunkSize(ilen);
		printf("original file length (padded): %zu\n", ilen);
		olen = ilen * 2;
		dlen = ilen;
		fseek(fp, 0, SEEK_SET);

#ifdef USE_UNIFIED_MEM
		hipMallocManaged(&in, ilen);
		hipMallocManaged(&compressed, olen);
		hipMallocManaged(&decompressed, dlen);
#else
		hipHostAlloc((void **)&in, ilen, hipHostMallocPortable);
		hipHostAlloc((void **)&compressed, olen,
			      hipHostMallocPortable);
		hipHostAlloc((void **)&decompressed, dlen,
			      hipHostMallocPortable);

		hipMalloc((void **)&compressedD, olen);
		hipMalloc((void **)&decompressedD, dlen);
		hipMemset(compressedD, 0, olen);
		hipMemset(decompressedD, 0, dlen);
#endif

		memset(in, 0, ilen);
		memset(compressed, 0, olen);
		memset(decompressed, 0, dlen);

		if (!fread(in, flen, 1, fp)) {
			fprintf(stderr,
				"FAIL: Reading file content to memory failed.\n");
		}
		fclose(fp);
	}

	if (ilen > LIB842_CUDA_CHUNK_SIZE) {
		printf("Using chunks of %d bytes\n", LIB842_CUDA_CHUNK_SIZE);

		size_t num_chunks = ilen / LIB842_CUDA_CHUNK_SIZE;
		size_t *compressedChunkPositions =
			(size_t *)malloc(sizeof(size_t) * num_chunks);
		size_t *compressedChunkSizes =
			(size_t *)malloc(sizeof(size_t) * num_chunks);

		timestart_comp = timestamp();
#pragma omp parallel for
		for (size_t chunk_num = 0; chunk_num < num_chunks;
		     chunk_num++) {
			size_t chunk_olen = LIB842_CUDA_CHUNK_SIZE * 2;
			uint8_t *chunk_in =
				in + (LIB842_CUDA_CHUNK_SIZE * chunk_num);
			uint8_t *chunk_out =
				compressed +
				((LIB842_CUDA_CHUNK_SIZE * 2) * chunk_num);

			optsw842_compress(chunk_in, LIB842_CUDA_CHUNK_SIZE, chunk_out,
				          &chunk_olen);
			compressedChunkSizes[chunk_num] = chunk_olen;
		}
		timeend_comp = timestamp();

		printf("Threads per Block: %d\n", THREADS_PER_BLOCK);

#if defined USE_STREAMS
		const size_t chunks_per_kernel =
			CHUNKS_PER_THREAD * THREADS_PER_BLOCK;
		int stream_counter = 0;
		timestart_decomp = timestamp();
		for (size_t i = 0; i < num_chunks; i += chunks_per_kernel) {
			hipMemcpyAsync(
				compressedD +
					((i * LIB842_CUDA_CHUNK_SIZE * 2) / 8),
				compressedH + (i * LIB842_CUDA_CHUNK_SIZE * 2),
				chunks_per_kernel * LIB842_CUDA_CHUNK_SIZE * 2,
				hipMemcpyHostToDevice,
				streams[stream_counter % STREAM_COUNT]);
			cuda842_decompress<<<
				chunks_per_kernel / THREADS_PER_BLOCK,
				THREADS_PER_BLOCK, 0,
				streams[stream_counter % STREAM_COUNT]>>>(
				compressedD +
					(i * (LIB842_CUDA_CHUNK_SIZE / 8) * 2),
				decompressedD + (i * (LIB842_CUDA_CHUNK_SIZE / 8)));
			hipMemcpyAsync(
				decompressedH + (i * LIB842_CUDA_CHUNK_SIZE),
				decompressedD + (i * (LIB842_CUDA_CHUNK_SIZE / 8)),
				chunks_per_kernel * LIB842_CUDA_CHUNK_SIZE,
				hipMemcpyDeviceToHost,
				streams[stream_counter % STREAM_COUNT]);
			stream_counter++;
		}
		hipDeviceSynchronize();
		cuda_error = hipGetLastError();
		CHECK_ERROR(cuda_error);
		timeend_decomp = timestamp();
#elif defined USE_UNIFIED_MEM
		const size_t chunks_per_kernel =
			CHUNKS_PER_THREAD * THREADS_PER_BLOCK;
		timestart_decomp = timestamp();
		for (size_t i = 0; i < num_chunks; i += chunks_per_kernel) {
			cuda842_decompress<<<chunks_per_kernel / THREADS_PER_BLOCK,
					     THREADS_PER_BLOCK, 0>>>(
				((uint64_t *)compressed) +
					(i * (LIB842_CUDA_CHUNK_SIZE / 8) * 2),
				((uint64_t *)decompressed) +
					(i * (LIB842_CUDA_CHUNK_SIZE / 8)));
		}
		hipDeviceSynchronize();
		cuda_error = hipGetLastError();
		CHECK_ERROR(cuda_error);
		timeend_decomp = timestamp();
#else
		cuda_error = hipMemcpy(compressedD, compressed, olen,
					hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		CHECK_ERROR(cuda_error);

		timestart_decomp = timestamp();
		cuda842_decompress<<<num_chunks / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(compressedD, decompressedD);
		hipDeviceSynchronize();
		cuda_error = hipGetLastError();
		CHECK_ERROR(cuda_error);
		timeend_decomp = timestamp();

		cuda_error = hipMemcpy(decompressed, decompressedD, dlen,
					hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		CHECK_ERROR(cuda_error);
#endif

		printf("Compression performance: %lld ms / %f MiB/s\n",
		       timeend_comp - timestart_comp,
		       (ilen / 1024 / 1024) /
			       ((float)(timeend_comp - timestart_comp) / 1000));
		printf("Decompression performance: %lld ms / %f MiB/s\n",
		       timeend_decomp - timestart_decomp,
		       (ilen / 1024 / 1024) /
			       ((float)(timeend_decomp - timestart_decomp) /
				1000));

	} else {
		optsw842_compress(in, ilen, compressed, &olen);
#ifdef USE_UNIFIED_MEM
		cuda842_decompress<<<1, 1>>>((uint64_t *)compressed,
					       (uint64_t *)decompressed);
		hipDeviceSynchronize();
		cuda_error = hipGetLastError();
		CHECK_ERROR(cuda_error);
#else
		cuda_error = hipMemcpy(compressedD, compressed, olen,
					hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		CHECK_ERROR(cuda_error);
		cuda842_decompress<<<1, 1>>>(compressedD, decompressedD);
		hipMemcpy(decompressed, decompressedD, dlen,
			   hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		cuda_error = hipGetLastError();
		CHECK_ERROR(cuda_error);
#endif
	}

	if (memcmp(in, decompressed, ilen) == 0) {
		printf("Compression- and decompression was successful!\n");
	} else {
		fprintf(stderr,
			"FAIL: Decompressed data differs from the original input data.\n");
	}

#ifdef USE_UNIFIED_MEM
	hipFree(in);
	hipFree(compressed);
	hipFree(decompressed);
#else
	hipHostFree(in);
	hipHostFree(compressed);
	hipHostFree(decompressed);

	hipFree(compressedD);
	hipFree(decompressedD);
#endif
#ifdef USE_STREAMS
	for (int i = 1; i < STREAM_COUNT; i++) {
		hipStreamDestroy(streams[i]);
	}
#endif
	printf("\n\n");
	return 0;
}
