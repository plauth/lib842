#include "hip/hip_runtime.h"
#include "842-internal.h"

struct sw842_param_decomp {
	uint64_t *out;
	const uint64_t *ostart;
	const uint64_t *in;
	uint32_t bits;
	uint64_t buffer;
};

/* number of bits in a buffered word */
#define WSIZE 64 //sizeof(uint64_t)

#ifdef LIB842_CUDA_STRICT
/* rolling fifo sizes */
#define I2_FIFO_SIZE (2 * (1 << I2_BITS))
#define I4_FIFO_SIZE (4 * (1 << I4_BITS))
#define I8_FIFO_SIZE (8 * (1 << I8_BITS))

#define __round_mask(x, y) ((__typeof__(x))((y)-1))
#define round_down(x, y) ((x) & ~__round_mask(x, y))

__constant__ static const uint16_t fifo_sizes[9] = { 0, 0, I2_FIFO_SIZE, 0, I4_FIFO_SIZE, 0,
						     0, 0, I8_FIFO_SIZE };
#endif

__constant__ static const uint8_t dec_templates[26][4][2] = {
	// params size in bits
	{ OP_DEC_D8, OP_DEC_N0, OP_DEC_N0, OP_DEC_N0 }, // 0x00: { D8, N0, N0, N0 }, 64 bits
	{ OP_DEC_D4, OP_DEC_D2, OP_DEC_I2, OP_DEC_N0 }, // 0x01: { D4, D2, I2, N0 }, 56 bits
	{ OP_DEC_D4, OP_DEC_I2, OP_DEC_D2, OP_DEC_N0 }, // 0x02: { D4, I2, D2, N0 }, 56 bits
	{ OP_DEC_D4, OP_DEC_I2, OP_DEC_I2, OP_DEC_N0 }, // 0x03: { D4, I2, I2, N0 }, 48 bits

	{ OP_DEC_D4, OP_DEC_I4, OP_DEC_N0, OP_DEC_N0 }, // 0x04: { D4, I4, N0, N0 }, 41 bits
	{ OP_DEC_D2, OP_DEC_I2, OP_DEC_D4, OP_DEC_N0 }, // 0x05: { D2, I2, D4, N0 }, 56 bits
	{ OP_DEC_D2, OP_DEC_I2, OP_DEC_D2, OP_DEC_I2 }, // 0x06: { D2, I2, D2, I2 }, 48 bits
	{ OP_DEC_D2, OP_DEC_I2, OP_DEC_I2, OP_DEC_D2 }, // 0x07: { D2, I2, I2, D2 }, 48 bits

	{ OP_DEC_D2, OP_DEC_I2, OP_DEC_I2, OP_DEC_I2 }, // 0x08: { D2, I2, I2, I2 }, 40 bits
	{ OP_DEC_D2, OP_DEC_I2, OP_DEC_I4, OP_DEC_N0 }, // 0x09: { D2, I2, I4, N0 }, 33 bits
	{ OP_DEC_I2, OP_DEC_D2, OP_DEC_D4, OP_DEC_N0 }, // 0x0a: { I2, D2, D4, N0 }, 56 bits
	{ OP_DEC_I2, OP_DEC_D4, OP_DEC_I2, OP_DEC_N0 }, // 0x0b: { I2, D4, I2, N0 }, 48 bits

	{ OP_DEC_I2, OP_DEC_D2, OP_DEC_I2, OP_DEC_D2 }, // 0x0c: { I2, D2, I2, D2 }, 48 bits
	{ OP_DEC_I2, OP_DEC_D2, OP_DEC_I2, OP_DEC_I2 }, // 0x0d: { I2, D2, I2, I2 }, 40 bits
	{ OP_DEC_I2, OP_DEC_D2, OP_DEC_I4, OP_DEC_N0 }, // 0x0e: { I2, D2, I4, N0 }, 33 bits
	{ OP_DEC_I2, OP_DEC_I2, OP_DEC_D4, OP_DEC_N0 }, // 0x0f: { I2, I2, D4, N0 }, 48 bits

	{ OP_DEC_I2, OP_DEC_I2, OP_DEC_D2, OP_DEC_I2 }, // 0x10: { I2, I2, D2, I2 }, 40 bits
	{ OP_DEC_I2, OP_DEC_I2, OP_DEC_I2, OP_DEC_D2 }, // 0x11: { I2, I2, I2, D2 }, 40 bits
	{ OP_DEC_I2, OP_DEC_I2, OP_DEC_I2, OP_DEC_I2 }, // 0x12: { I2, I2, I2, I2 }, 32 bits
	{ OP_DEC_I2, OP_DEC_I2, OP_DEC_I4, OP_DEC_N0 }, // 0x13: { I2, I2, I4, N0 }, 25 bits

	{ OP_DEC_I4, OP_DEC_D4, OP_DEC_N0, OP_DEC_N0 }, // 0x14: { I4, D4, N0, N0 }, 41 bits
	{ OP_DEC_I4, OP_DEC_D2, OP_DEC_I2, OP_DEC_N0 }, // 0x15: { I4, D2, I2, N0 }, 33 bits
	{ OP_DEC_I4, OP_DEC_I2, OP_DEC_D2, OP_DEC_N0 }, // 0x16: { I4, I2, D2, N0 }, 33 bits
	{ OP_DEC_I4, OP_DEC_I2, OP_DEC_I2, OP_DEC_N0 }, // 0x17: { I4, I2, I2, N0 }, 25 bits

	{ OP_DEC_I4, OP_DEC_I4, OP_DEC_N0, OP_DEC_N0 }, // 0x18: { I4, I4, N0, N0 }, 18 bits
	{ OP_DEC_I8, OP_DEC_N0, OP_DEC_N0, OP_DEC_N0 }, // 0x19: { I8, N0, N0, N0 }, 8 bits
};

__device__ static inline uint64_t bswap(uint64_t value)
{
	asm("{\n\t"
	    "		.reg .b32 %li,%lo,%hi,%ho;\n\t"
	    "		mov.b64 {%li,%hi}, %0;\n\t"
	    "		prmt.b32 %lo, %li, %hi, 0x4567;\n\t"
	    " 		prmt.b32 %ho, %li, %hi, 0x0123;\n\t"
	    "		mov.b64 %0, {%lo,%ho};\n\t"
	    "}"
	    : "+l"(value));
	return value;
}

__device__ static inline uint64_t read_bits(struct sw842_param_decomp *p, uint32_t n)
{
	uint64_t value = p->buffer >> (WSIZE - n);
	//value = 0; if (n <= 0)
	asm("{\n\t"
	    "		.reg .pred %p;\n\t"
	    "		setp.ls.u32 %p, %1, 0;\n\t"
	    "@%p	mov.u64 %0, 0;\n\t"
	    "}"
	    : "+l"(value)
	    : "r"(n));

	if (p->bits < n) {
		p->buffer = bswap(*p->in);
		p->in++;
		value |= p->buffer >> (WSIZE - (n - p->bits));
		p->buffer <<= n - p->bits;
		p->bits += WSIZE - n;
		p->buffer *= (p->bits > 0);
	} else {
		p->bits -= n;
		p->buffer <<= n;
	}

	return value;
}

#ifdef LIB842_CUDA_STRICT
__device__ static inline uint64_t get_index(const struct sw842_param_decomp *p,
					    uint8_t size,
					    uint64_t index, uint64_t fsize)
{
	uint64_t offset;
	uint64_t total = round_down(
		((uint8_t *)p->out) - ((const uint8_t *)p->ostart), 8);

	offset = index * size;

	/* a ring buffer of fsize is used; correct the offset */
	if (total > fsize) {
		/* this is where the current fifo is */
		uint64_t section = round_down(total, fsize);
		/* the current pos in the fifo */
		uint64_t pos = total - section;

		/* if the offset is past/at the pos, we need to
		 * go back to the last fifo section
		 */
		if (offset >= pos)
			section -= fsize;

		offset += section;
	}

	return offset;
}
#endif

__global__ void cuda842_decompress(__restrict__ const uint64_t *in,
				   __restrict__ uint64_t *out)
{
	unsigned int chunk_num = blockIdx.x * blockDim.x + threadIdx.x;

	struct sw842_param_decomp p;
	p.ostart = p.out = out + ((LIB842_CUDA_CHUNK_SIZE / 8) * chunk_num);
	p.in = (in + ((LIB842_CUDA_CHUNK_SIZE / 8 * 2) * chunk_num));

	p.buffer = 0;
	p.bits = 0;

	uint64_t op;

	uint64_t output_word;
	uint32_t bits;

#ifdef LIB842_CUDA_STRICT
	do {
		op = read_bits(&p, OP_BITS);

		output_word = 0;
		bits = 0;

		switch (op) {
		case OP_REPEAT:
			op = read_bits(&p, REPEAT_BITS);
			/* copy op + 1 */
			op++;

			while (op-- > 0) {
				*p.out = *(p.out - 1);
				p.out++;
			}
			break;
		case OP_ZEROS:
			*p.out = 0;
			p.out++;
			break;
		case OP_END:
			break;
		default:
#else
	while (op = read_bits(&p, OP_BITS), op != OP_END) {
		output_word = 0;
		bits = 0;
#endif
			for (int i = 0; i < 4; i++) {
				uint64_t value;

				uint32_t dec_template = dec_templates[op][i][0];
				uint32_t is_index = (dec_template >> 7);
				uint32_t dst_size = dec_templates[op][i][1];

				value = read_bits(&p, dec_template & 0x7F);
#ifdef LIB842_CUDA_STRICT
				if (is_index) {
					uint64_t offset =
						get_index(&p, dst_size, value,
							  fifo_sizes[dst_size]);

					asm("{\n\t"
					    "		.reg .pred %pr4, %pr8;\n\t"
					    "		.reg .u16 %val16_0, %val16_1, %val16_2, %val16_3;\n\t"
					    "		.reg .u32 %val32;\n\t"
					    "		.reg .u64 %addr, %result;\n\t"

					    "		setp.hi.u32 %pr4, %2, 2;\n\t"
					    "		setp.eq.u32 %pr8, %2, 8;\n\t"

					    "		add.u64 %addr, %1, %3;\n\t"
					    "		ld.global.b16 %val16_0, [%addr];\n\t"
					    "@%pr4	ld.global.b16 %val16_1, [%addr+2];\n\t"
					    "@%pr8	ld.global.b16 %val16_2, [%addr+4];\n\t"
					    "@%pr8	ld.global.b16 %val16_3, [%addr+6];\n\t"
					    "		cvt.u64.u16 %result, %val16_0;\n\t"
					    "@%pr4	mov.b32 %val32, {%val16_0, %val16_1};\n\t"
					    "@%pr4	cvt.u64.u32 %result, %val32;\n\t"
					    "@%pr8	mov.b64 %result, {%val16_0, %val16_1, %val16_2, %val16_3};\n\t"
					    "		shl.b32 %val32, %2, 3;\n\t"
					    "		sub.u32 %val32, 64, %val32;\n\t"
					    "		shl.b64 %result, %result, %val32;\n\t"

					    "		.reg .b32 %li,%lo,%hi,%ho;\n\t"
					    "		mov.b64 {%li,%hi}, %result;\n\t"
					    "		prmt.b32 %lo, %li, %hi, 0x4567;\n\t"
					    " 		prmt.b32 %ho, %li, %hi, 0x0123;\n\t"
					    "		mov.b64 %result, {%lo,%ho};\n\t"
					    "		mov.b64 %0, %result;\n\t"

					    "}"
					    : "+l"(value)
					    : "l"(p.ostart), "r"(dst_size),
					      "l"(offset)

					);
				}
				output_word |= value
					       << (64 - (dst_size << 3) - bits);
				bits += dst_size << 3;
#else
			asm("{\n\t"
			    "		.reg .pred %pr2, %pr4, %pr8, %pi;\n\t"
			    "		.reg .u16 %val16_0, %val16_1, %val16_2, %val16_3;\n\t"
			    "		.reg .u32 %val32, %nbits;\n\t"
			    "		.reg .u64 %addr, %result;\n\t"

			    "		setp.eq.u32 %pi, %4, 1;\n\t"
			    "@%pi	setp.hs.u32 %pr2, %3, 2;\n\t"
			    "@%pi	setp.hi.u32 %pr4, %3, 2;\n\t"
			    "@%pi	setp.eq.u32 %pr8, %3, 8;\n\t"
			    "@!%pi	setp.eq.u32 %pr2, 0, 1;\n\t"
			    "@!%pi	setp.eq.u32 %pr4, 0, 1;\n\t"
			    "@!%pi	setp.eq.u32 %pr8, 0, 1;\n\t"

			    "		cvt.u64.u32 %addr, %3;\n\t"
			    "		mul.lo.u64 %addr, %5, %addr;\n\t"
			    "		add.u64 %addr, %addr, %2;\n\t"

			    "@%pr2	ld.global.b16 %val16_0, [%addr];\n\t"
			    "@%pr4	ld.global.b16 %val16_1, [%addr+2];\n\t"
			    "@%pr8	ld.global.b16 %val16_2, [%addr+4];\n\t"
			    "@%pr8	ld.global.b16 %val16_3, [%addr+6];\n\t"
			    "		cvt.u64.u16 %result, %val16_0;\n\t"
			    "@%pr4	mov.b32 %val32, {%val16_0, %val16_1};\n\t"
			    "@%pr4	cvt.u64.u32 %result, %val32;\n\t"
			    "@%pr8	mov.b64 %result, {%val16_0, %val16_1, %val16_2, %val16_3};\n\t"

			    "		shl.b32 %nbits, %3, 3;\n\t"
			    "		sub.u32 %val32, 64, %nbits;\n\t"
			    "		shl.b64 %result, %result, %val32;\n\t"

			    "		.reg .b32 %li,%lo,%hi,%ho;\n\t"
			    "		mov.b64 {%li,%hi}, %result;\n\t"
			    "		prmt.b32 %lo, %li, %hi, 0x4567;\n\t"
			    " 		prmt.b32 %ho, %li, %hi, 0x0123;\n\t"
			    "		mov.b64 %result, {%lo,%ho};\n\t"
			    "@%pi	mov.b64 %5, %result;\n\t"

			    "		sub.u32 %val32, %val32, %1;\n\t"
			    "		shl.b64 %5, %5, %val32;\n\t"
			    "		or.b64 %0, %0, %5;\n\t"

			    "		add.u32 %1, %1, %nbits;\n\t"

			    "}"

			    : "+l"(output_word), "+r"(bits)
			    : "l"(p.ostart), "r"(dst_size), "r"(is_index),
			      "l"(value)

			);
#endif
			}
			*p.out++ = bswap(output_word);

#ifdef LIB842_CUDA_STRICT
		}
	} while (op != OP_END);
#else
	}
#endif

	return;
}
